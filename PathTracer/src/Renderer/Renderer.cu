#include "hip/hip_runtime.h"
#include "Renderer.cuh"
#include "../Utils.h"

__global__ void traceRay(void *device_ptr, uint32_t imageWidth, uint32_t imageHeight)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float x = (float)i / (float)imageWidth * 2.0f - 1.0f;
	float y = (float)j / (float)imageHeight * 2.0f - 1.0f;

	if (i >= imageWidth || j >= imageHeight)
		return;

	uint32_t* imagePtr = (uint32_t*)device_ptr;

	glm::vec3 rayOrigin(0, 0, 2.0f);
	glm::vec3 rayDirection(x, y, -1.0f);

	float radius = 0.5f;

	float a = glm::dot(rayDirection, rayDirection);
	float b = 2.0f * glm::dot(rayOrigin, rayDirection);
	float c = glm::dot(rayOrigin, rayOrigin) - radius * radius;


	float discriminant = b * b - 4.0f * a * c;

	if (discriminant < 0.0f)
	{
		imagePtr[j * imageWidth + i] = 0xff000000;
		return;
	}

	float t0 = (- b + glm::sqrt(discriminant)) / 2.0f * a;
	float t1 = (- b - glm::sqrt(discriminant)) / 2.0f * a;

	glm::vec3 hitPoint = rayOrigin + rayDirection * t1;
	glm::vec3 normal = glm::normalize(hitPoint);

	glm::vec3 lightDir = glm::normalize(glm::vec3(-1.0f, -1.0f, -1.0f));

	float d = glm::max(glm::dot(normal, -lightDir), 0.0f);

	glm::vec3 sphereColor(1.0f, 0.0f, 1.0f);
	sphereColor = sphereColor * d;

	glm::vec4 color = glm::clamp(glm::vec4(sphereColor, 1.0f), glm::vec4(0.0f), glm::vec4(1.0f));
	uint8_t red = (uint8_t)(color.r * 255.0f);
	uint8_t green = (uint8_t)(color.g * 255.0f);
	uint8_t blue = (uint8_t)(color.b * 255.0f);
	uint8_t alpha = (uint8_t)(color.a * 255.0f);
	 
	imagePtr[j * imageWidth + i] = alpha << 24 | blue << 16 | green << 8 | red;

}

void RenderViewport(std::shared_ptr<PixelBuffer> pixelBuffer)
{
	checkCudaErrors(hipGraphicsMapResources(1, &pixelBuffer->GetCudaResource()));
	size_t size = 0;
	void* devicePtr = 0;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer(&devicePtr, &size, pixelBuffer->GetCudaResource()));

	uint32_t tx = 8, ty = 8;
	dim3 blocks(pixelBuffer->GetWidth() / tx + 1, pixelBuffer->GetHeight() / ty + 1);
	dim3 threads(tx, ty);

	traceRay<<<blocks, threads>>>(devicePtr, pixelBuffer->GetWidth(), pixelBuffer->GetHeight());

	checkCudaErrors(hipGraphicsUnmapResources(1, &pixelBuffer->GetCudaResource(), 0));
}

