#include "hip/hip_runtime.h"
#include "Renderer.cuh"
#include "cuda/cuda_math.h"
#include "../Utils/Utils.h"
#include "../Camera.h"

__device__ __constant__ CameraData cameraData;
__device__ __constant__ SceneData sceneData;


__global__ void traceRay(void *bufferDevicePtr)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float x = i / (float)cameraData.viewportWidth;
	float y = j / (float)cameraData.viewportHeight;

	if (i >= cameraData.viewportWidth || j >= cameraData.viewportHeight)
		return;

	uint32_t* imagePtr = (uint32_t*)bufferDevicePtr;

	Ray ray(
		cameraData.position,
		cameraData.lowerLeftCorner + x * cameraData.horizontal + y * cameraData.vertical - cameraData.position
	);

	Sphere* closestSphere = nullptr;
	float hitDistance = FLT_MAX;
	HitResult hitResult;

	for (int i = 0; i < sceneData.nSpheres; i++)
	{
		if (sceneData.spheres[i].Hit(ray, hitResult))
		{
			hitDistance = hitResult.t;
			closestSphere = &sceneData.spheres[i];
		}

		//float3 origin = ray.origin - sceneData.spheres[i].position;

		//float radius = sceneData.spheres[i].radius;

		//float a = dot(ray.direction, ray.direction);
		//float b = dot(origin, ray.direction);
		//float c = dot(origin, origin) - radius * radius;

		//float discriminant = b * b - a * c;

		//if (discriminant < 0.0f)
		//{
		//	imagePtr[j * cameraData.viewportWidth + i] = 0xff000000;
		//	continue;
		//}

		//float closestT = (-b - sqrt(discriminant)) / a;

		//if (closestT < hitDistance && closestT > 0.0f)
		//{
		//	hitDistance = closestT;
		//	closestSphere = &sceneData.spheres[i];
		//}
	}

	if (closestSphere == nullptr)
	{
		imagePtr[j * cameraData.viewportWidth + i] = 0xff000000;
		return;
	}

	float3 hitPoint = ray.origin + ray.direction * hitDistance;
	float3 normal = (hitPoint - closestSphere->position) / closestSphere->radius;

	float3 lightDir = normalize(make_float3(-1.0f));

	float d = max(dot(normal, -lightDir), 0.0f);

	float3 sphereColor = closestSphere->material->color;
	sphereColor = sphereColor * d;

	float4 color = clamp(make_float4(sphereColor, 1.0f), make_float4(0.0f), make_float4(1.0f));
	uint8_t red = (uint8_t)(color.x * 255.0f);
	uint8_t green = (uint8_t)(color.y * 255.0f);
	uint8_t blue = (uint8_t)(color.z * 255.0f);
	uint8_t alpha = (uint8_t)(color.w * 255.0f);
	 
	imagePtr[j * cameraData.viewportWidth + i] = alpha << 24 | blue << 16 | green << 8 | red;

}

void RenderViewport(std::shared_ptr<PixelBuffer> pixelBuffer)
{
	checkCudaErrors(hipGraphicsMapResources(1, &pixelBuffer->GetCudaResource()));
	size_t size = 0;
	void* devicePtr = 0;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer(&devicePtr, &size, pixelBuffer->GetCudaResource()));

	uint32_t tx = 16, ty = 16;
	dim3 blocks(pixelBuffer->GetWidth() / tx + 1, pixelBuffer->GetHeight() / ty + 1);
	dim3 threads(tx, ty);

	traceRay<<<blocks, threads>>>(devicePtr);

	checkCudaErrors(hipGraphicsUnmapResources(1, &pixelBuffer->GetCudaResource(), 0));
}

void SendCameraDataToDevice(Camera* camera)
{
	float3 position = camera->GetPosition();
	float3 forwardDirection = camera->GetForwardDirection();
	float3 rightDirection = camera->GetRightDirection();
	float3 upDirection = cross(rightDirection, forwardDirection);

	float aspectRatio = camera->GetViewportWidth() / (float)camera->GetViewportHeight();
	float halfHeight = tanf(camera->GetVerticalFOV() / 2.0f * M_PI / 180.0f);
	float halfWidth = aspectRatio * halfHeight;

	float3 lowerLeftCorner = position - halfWidth * rightDirection - halfHeight * upDirection + forwardDirection;
	float3 horizontal = 2 * halfWidth * rightDirection;
	float3 vertical = 2 * halfHeight * upDirection;

	CameraData data = {
		position,
		forwardDirection,
		lowerLeftCorner,
		horizontal,
		vertical,
		camera->GetViewportWidth(),
		camera->GetViewportHeight()
	};
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cameraData), &data, sizeof(CameraData)));
}

void SendSceneDataToDevice(Scene* scene)
{
	SceneData data;
	std::vector<Sphere> spheres = scene->GetSpheres();
	data.nSpheres = spheres.size();
	for (int i = 0; i < spheres.size(); i++)
	{
		data.spheres[i] = spheres[i];
	}
	// TODO: change the size of copy
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sceneData), &data, sizeof(unsigned int) + sizeof(Sphere) * data.nSpheres));
}
