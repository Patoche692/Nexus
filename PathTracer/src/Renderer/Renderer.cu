#include "hip/hip_runtime.h"
#include "Renderer.cuh"
#include "cuda/cuda_math.h"
#include "../Utils.h"

__device__ __constant__ CameraData cameraData;
__device__ __constant__ SceneData sceneData;


__global__ void traceRay(void *bufferDevicePtr)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float x = (float)i / (float)cameraData.viewportWidth;
	float y = (float)j / (float)cameraData.viewportHeight;

	if (i >= cameraData.viewportWidth || j >= cameraData.viewportHeight)
		return;

	uint32_t* imagePtr = (uint32_t*)bufferDevicePtr;

	float3 rayOrigin = cameraData.position;
	float3 origin = rayOrigin - sceneData.spheres[0].position;
	float3 rayDirection = normalize(cameraData.lowerLeftCorner + x * cameraData.horizontal + y * cameraData.vertical - rayOrigin);

	float radius = sceneData.spheres[0].radius;

	float a = dot(rayDirection, rayDirection);
	float b = 2.0f * dot(origin, rayDirection);
	float c = dot(origin, origin) - radius * radius;


	float discriminant = b * b - 4.0f * a * c;

	float closestT = (- b - sqrt(discriminant)) / 2.0f * a;

	if (discriminant < 0.0f || closestT < 0.0f)
	{
		imagePtr[j * cameraData.viewportWidth + i] = 0xff000000;
		return;
	}

	float3 hitPoint = origin + rayDirection * closestT;
	float3 normal = normalize(hitPoint);

	float3 lightDir = normalize(make_float3(-1.0f, -1.0f, -1.0f));

	float d = max(dot(normal, -lightDir), 0.0f);

	float3 sphereColor = sceneData.spheres[0].material.color;
	sphereColor = sphereColor * d;

	float4 color = clamp(make_float4(sphereColor, 1.0f), make_float4(0.0f), make_float4(1.0f));
	uint8_t red = (uint8_t)(color.x * 255.0f);
	uint8_t green = (uint8_t)(color.y * 255.0f);
	uint8_t blue = (uint8_t)(color.z * 255.0f);
	uint8_t alpha = (uint8_t)(color.w * 255.0f);
	 
	imagePtr[j * cameraData.viewportWidth + i] = alpha << 24 | blue << 16 | green << 8 | red;

}

void RenderViewport(std::shared_ptr<PixelBuffer> pixelBuffer)
{
	checkCudaErrors(hipGraphicsMapResources(1, &pixelBuffer->GetCudaResource()));
	size_t size = 0;
	void* devicePtr = 0;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer(&devicePtr, &size, pixelBuffer->GetCudaResource()));

	uint32_t tx = 8, ty = 8;
	dim3 blocks(pixelBuffer->GetWidth() / tx + 1, pixelBuffer->GetHeight() / ty + 1);
	dim3 threads(tx, ty);

	traceRay<<<blocks, threads>>>(devicePtr);

	checkCudaErrors(hipGraphicsUnmapResources(1, &pixelBuffer->GetCudaResource(), 0));
}

void SendCameraDataToDevice(Camera* camera)
{
	glm::vec3 position = camera->GetPosition();
	glm::vec3 forwardDirection = camera->GetForwardDirection();
	glm::vec3 rightDirection = camera->GetRightDirection();
	glm::vec3 upDirection = glm::cross(rightDirection, forwardDirection);

	float aspectRatio = camera->GetViewportWidth() / (float)camera->GetViewportHeight();
	float halfHeight = tanf(camera->GetVerticalFOV() / 2.0f * M_PI / 180.0f);
	float halfWidth = aspectRatio * halfHeight;

	glm::vec3 lowerLeftCorner = position - halfWidth * rightDirection - halfHeight * upDirection + forwardDirection;
	glm::vec3 horizontal = 2 * halfWidth * rightDirection;
	glm::vec3 vertical = 2 * halfHeight * upDirection;

	CameraData data = {
		make_float3(position),
		make_float3(forwardDirection),
		make_float3(lowerLeftCorner),
		make_float3(horizontal),
		make_float3(vertical),
		camera->GetViewportWidth(),
		camera->GetViewportHeight()
	};
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cameraData), &data, sizeof(CameraData)));
}

void SendSceneDataToDevice(Scene* scene)
{
	SceneData data;
	std::vector<Sphere> spheres = scene->GetSpheres();
	for (int i = 0; i < spheres.size(); i++)
	{
		data.spheres[i] = {
			spheres[i].radius,
			make_float3(spheres[i].position),
			{ make_float3(spheres[i].material.color) }
		};
	}
	// TODO: change the size of copy
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sceneData), &data, sizeof(Sphere) * spheres.size()));
}
