#include "hip/hip_runtime.h"
#include "PathTracer.cuh"
#include "Random.cuh"
#include "BRDF.cuh"
#include "BSDF/DielectricBSDF.cuh"
#include "Utils/cuda_math.h"
#include "Utils/Utils.h"
#include "Camera.h"
#include "Geometry/BVH/TLAS.h"


__constant__ __device__ CameraData cameraData;
extern __constant__ __device__ Material* materials;
extern __constant__ __device__ Texture* textures;
extern __constant__ __device__ Mesh* bvhs;
extern __constant__ __device__ TLAS tlas;

inline __device__ uint32_t toColorUInt(float3 color)
{
	float4 clamped = clamp(make_float4(color, 1.0f), make_float4(0.0f), make_float4(1.0f));
	uint8_t red = (uint8_t)(clamped.x * 255.0f);
	uint8_t green = (uint8_t)(clamped.y * 255.0f);
	uint8_t blue = (uint8_t)(clamped.z * 255.0f);
	uint8_t alpha = (uint8_t)(clamped.w * 255.0f);
	 
	return alpha << 24 | blue << 16 | green << 8 | red;
}

inline __device__ float3 color(Ray& r, unsigned int& rngState)
{
	Ray currentRay = r;
	float3 currentThroughput = make_float3(1.0f);
	float3 emission = make_float3(0.0f);

	for (int j = 0; j < 8; j++)
	{
		// Reset the hit position and calculate the inverse of the new direction
		currentRay.hit.t = 1e30f;
		currentRay.invDirection = 1 / currentRay.direction;

		tlas.Intersect(currentRay);

		// If no intersection, sample background
		if (currentRay.hit.t == 1e30f)
			return currentThroughput * make_float3(0.00f) + emission;

		HitResult hitResult;
		hitResult.p = currentRay.origin + currentRay.direction * currentRay.hit.t;
		hitResult.rIn = currentRay;

		BVHInstance& instance = tlas.blas[currentRay.hit.instanceIdx];
		Triangle& triangle = instance.bvh->triangles[currentRay.hit.triIdx];
		float u = currentRay.hit.u, v = currentRay.hit.v;

		// Interpolating and rotating the normal
		hitResult.normal = u * triangle.normal1 + v * triangle.normal2 + (1 - (u + v)) * triangle.normal0;
		hitResult.normal = normalize(instance.transform.TransformVector(hitResult.normal));

		hitResult.material = materials[instance.materialId];

		if (hitResult.material.diffuseMapId == -1)
			hitResult.albedo = hitResult.material.diffuse;
		else
		{
			float2 uv = u * triangle.texCoord1 + v * triangle.texCoord2 + (1 - (u + v)) * triangle.texCoord0;
			hitResult.material.diffuse = textures[hitResult.material.diffuseMapId].GetPixel(uv.x, uv.y);
		}
		// Normal flipping
		//if (dot(hitResult.normal, currentRay.direction) > 0.0f)
		//	hitResult.normal = -hitResult.normal;

		if (dot(hitResult.material.emissive, hitResult.material.emissive) > 0.0f)
			emission += hitResult.material.emissive * currentThroughput;


		// Transform the incoming ray to local space (positive Z axis aligned with shading normal)
		float4 qRotationToZ = getRotationToZAxis(hitResult.normal);
		float3 wi = rotatePoint(qRotationToZ, -hitResult.rIn.direction);

		float3 throughput;
		float3 wo;
		DielectricBSDF bsdf;
		bsdf.PrepareBSDFData(wi, hitResult.material);

		float3 gNormal = normalize(instance.transform.TransformVector(triangle.Normal()));
		if (bsdf.Eval(hitResult, wi, wo, throughput, rngState, gNormal))
		{
			float3 outDir = normalize(rotatePoint(invertRotation(qRotationToZ), wo));
			bool geometryBackSide = dot(outDir, gNormal) < 0.0f;
			// TODO: take into account instance rotation
			bool shadingBackSide = dot(outDir, hitResult.normal) < 0.0f;

			if (geometryBackSide == shadingBackSide)
			{
				currentThroughput *= throughput;
				float offsetDirection = geometryBackSide ? -1.0f : 1.0f;
				currentRay.origin = hitResult.p + offsetDirection * 1.0e-4 * hitResult.normal;
				// Inverse ray transformation to world space
				currentRay.direction = outDir;
			}
		}


		// Russian roulette
		float p = clamp(fmax(currentThroughput.x, fmax(currentThroughput.y, currentThroughput.z)), 0.01f, 1.0f);
		if (j > 2)
		{
			if (Random::Rand(rngState) < p)
			{
				// To get unbiased results, we need to increase the contribution of
				// the non-terminated rays with their probability of being terminated
				currentThroughput *= 1.0f / p;
			}
			else
				return emission;
		}
	}

	return emission;
}

__global__ void traceRay(uint32_t* outBufferPtr, uint32_t frameNumber, float3* accumulationBuffer)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	uint2 pixel = make_uint2(i, j);

	uint2 resolution = cameraData.resolution;

	if (pixel.x >= resolution.x || pixel.y >= resolution.y)
		return;

	unsigned int rngState = Random::InitRNG(pixel, resolution, frameNumber);

	// Avoid using modulo, it significantly impacts performance
	float x = (pixel.x + Random::Rand(rngState)) / (float)resolution.x;
	float y = (pixel.y + Random::Rand(rngState)) / (float)resolution.y;

	float2 rd = cameraData.lensRadius * Random::RandomInUnitDisk(rngState);
	float3 offset = cameraData.right * rd.x + cameraData.up * rd.y;

	Ray ray(
		cameraData.position + offset,
		normalize(cameraData.lowerLeftCorner + x * cameraData.viewportX + y * cameraData.viewportY - cameraData.position - offset)
	);

	float3 c = color(ray, rngState);	// get new color
	if (frameNumber == 1)
		accumulationBuffer[pixel.y * resolution.x + pixel.x] = c;
	else
		accumulationBuffer[pixel.y * resolution.x + pixel.x] += c;

	c = accumulationBuffer[pixel.y * resolution.x + pixel.x] / frameNumber;

	// Gamma correction
	c = make_float3(pow(c.x, 0.455), pow(c.y, 0.455), pow(c.z, 0.455));

	outBufferPtr[pixel.y * resolution.x + pixel.x] = toColorUInt(c);	// convert color
}

void RenderViewport(std::shared_ptr<PixelBuffer> pixelBuffer, uint32_t frameNumber, float3* accumulationBuffer)
{
	checkCudaErrors(hipGraphicsMapResources(1, &pixelBuffer->GetCudaResource()));
	size_t size = 0;
	uint32_t* devicePtr = 0;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&devicePtr, &size, pixelBuffer->GetCudaResource()));

	uint32_t tx = 16, ty = 16;
	dim3 blocks(pixelBuffer->GetWidth() / tx + 1, pixelBuffer->GetHeight() / ty + 1);
	dim3 threads(tx, ty);

	traceRay<<<blocks, threads>>>(devicePtr, frameNumber, accumulationBuffer);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipGraphicsUnmapResources(1, &pixelBuffer->GetCudaResource(), 0));
}

void SendCameraDataToDevice(Camera* camera)
{
	float3 position = camera->GetPosition();
	float3 forwardDirection = camera->GetForwardDirection();
	float3 rightDirection = camera->GetRightDirection();
	float3 upDirection = cross(rightDirection, forwardDirection);

	float aspectRatio = camera->GetViewportWidth() / (float)camera->GetViewportHeight();
	float halfHeight = camera->GetFocusDist() * tanf(camera->GetVerticalFOV() / 2.0f * M_PI / 180.0f);
	float halfWidth = aspectRatio * halfHeight;

	float3 viewportX = 2 * halfWidth * rightDirection;
	float3 viewportY = 2 * halfHeight * upDirection;
	float3 lowerLeftCorner = position - viewportX / 2.0f - viewportY / 2.0f + forwardDirection * camera->GetFocusDist();

	float lensRadius = camera->GetFocusDist() * tanf(camera->GetDefocusAngle() / 2.0f * M_PI / 180.0f);

	CameraData data = {
		position,
		rightDirection,
		upDirection,
		lensRadius,
		lowerLeftCorner,
		viewportX,
		viewportY,
		make_uint2(camera->GetViewportWidth(), camera->GetViewportHeight())
	};
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cameraData), &data, sizeof(CameraData)));
}