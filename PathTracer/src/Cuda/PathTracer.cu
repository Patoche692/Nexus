#include "hip/hip_runtime.h"
#include "PathTracer.cuh"
#include "Random.cuh"
#include "BRDF.cuh"
#include "Utils/cuda_math.h"
#include "Utils/Utils.h"
#include "Camera.h"
#include "Geometry/BVH/TLAS.h"


__device__ __constant__ CameraData cameraData;
extern __constant__ __device__ Material* materials;
extern __constant__ __device__ Texture* textures;
extern __constant__ __device__ Mesh* bvhs;
extern __constant__ __device__ TLAS tlas;

inline __device__ uint32_t toColorUInt(float3 color)
{
	float4 clamped = clamp(make_float4(color, 1.0f), make_float4(0.0f), make_float4(1.0f));
	uint8_t red = (uint8_t)(clamped.x * 255.0f);
	uint8_t green = (uint8_t)(clamped.y * 255.0f);
	uint8_t blue = (uint8_t)(clamped.z * 255.0f);
	uint8_t alpha = (uint8_t)(clamped.w * 255.0f);
	 
	return alpha << 24 | blue << 16 | green << 8 | red;
}

inline __device__ float3 color(Ray& r, unsigned int& rngState)
{
	Ray currentRay = r;
	float3 currentAttenuation = make_float3(1.0f);
	const float russianRouProb = 0.7f;				// low number = earlier break up

	for (int j = 0; j < 6; j++)
	{
		// Reset the hit position and calculate the inverse of the new direction
		currentRay.hit.t = 1e30f;
		currentRay.invDirection = 1 / currentRay.direction;

		tlas.Intersect(currentRay);
		if (currentRay.hit.t != 1e30f)
		{
			HitResult hitResult;
			hitResult.p = currentRay.origin + currentRay.direction * currentRay.hit.t;
			hitResult.rIn = currentRay;

			BVHInstance instance = tlas.blas[currentRay.hit.instanceIdx];
			Triangle& triangle = instance.bvh->triangles[currentRay.hit.triIdx];
			float u = currentRay.hit.u, v = currentRay.hit.v;

			// Interpolating and rotating the normal
			hitResult.normal = u * triangle.normal1 + v * triangle.normal2 + (1 - (u + v)) * triangle.normal0;
			hitResult.normal = normalize(instance.transform.TransformVector(hitResult.normal));

			hitResult.material = materials[instance.materialId];


			// Normal flipping
			//if (dot(hitResult.normal, currentRay.direction) > 0.0f)
			//	hitResult.normal = -hitResult.normal;

			float3 attenuation = make_float3(1.0f);
			Ray scatterRay = currentRay;
			
			switch (hitResult.material.type)
			{
			case Material::Type::DIFFUSE:
				if (hitResult.material.textureId == -1)
					hitResult.albedo = hitResult.material.diffuse.albedo;
				else
				{
					float2 uv = u * triangle.texCoord1 + v * triangle.texCoord2 + (1 - (u + v)) * triangle.texCoord0;
					hitResult.albedo = textures[hitResult.material.textureId].GetPixel(uv.x, uv.y);
				}

				if (diffuseScatter(hitResult, attenuation, scatterRay, rngState))
				{
					currentAttenuation *= attenuation;
					currentRay = scatterRay;
				}
				break;
			case Material::Type::METAL:
				if (hitResult.material.textureId == -1)
					hitResult.albedo = hitResult.material.diffuse.albedo;
				else
				{
					float2 uv = u * triangle.texCoord1 + v * triangle.texCoord2 + (1 - (u + v)) * triangle.texCoord0;
					hitResult.albedo = textures[hitResult.material.textureId].GetPixel(uv.x, uv.y);
				}
				if (plasticScattter(hitResult, attenuation, scatterRay, rngState))
				{
					currentAttenuation *= attenuation;
					currentRay = scatterRay;
				}
				break;
			case Material::Type::DIELECTRIC:
				if (dielectricScattter(hitResult, attenuation, scatterRay, rngState))
				{
					currentAttenuation *= attenuation;
					currentRay = scatterRay;
				}
				break;
			case Material::Type::LIGHT:
				return currentAttenuation * hitResult.material.light.emission;
				break;
			default:
				break;
			}

			//float randNr = Random::Rand(rngState);
			//if (randNr > russianRouProb ? true : (j > 0 ? (currentAttenuation /= russianRouProb, false) : false)) break;
		}
		else
			return currentAttenuation * make_float3(0.2f);
	}

	return currentAttenuation * make_float3(0.2f);
}

__global__ void traceRay(uint32_t* outBufferPtr, uint32_t frameNumber, float3* accumulationBuffer)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	uint2 pixel = make_uint2(i, j);

	uint2 resolution = cameraData.resolution;

	if (pixel.x >= resolution.x || pixel.y >= resolution.y)
		return;

	unsigned int rngState = Random::InitRNG(pixel, resolution, frameNumber);

	// Avoid using modulo, it significantly impacts performance
	float x = (pixel.x + Random::Rand(rngState)) / (float)resolution.x;
	float y = (pixel.y + Random::Rand(rngState)) / (float)resolution.y;

	float2 rd = cameraData.lensRadius * Random::RandomInUnitDisk(rngState);
	float3 offset = cameraData.right * rd.x + cameraData.up * rd.y;

	Ray ray(
		cameraData.position + offset,
		normalize(cameraData.lowerLeftCorner + x * cameraData.viewportX + y * cameraData.viewportY - cameraData.position - offset)
	);

	float3 c = color(ray, rngState);									// get new colour
	if (frameNumber == 1)
		accumulationBuffer[pixel.y * resolution.x + pixel.x] = c;
	else
		accumulationBuffer[pixel.y * resolution.x + pixel.x] += c;

	c = accumulationBuffer[pixel.y * resolution.x + pixel.x] / frameNumber;

	// Gamma correction
	//c = make_float3(sqrt(c.x), sqrt(c.y), sqrt(c.z));
	outBufferPtr[pixel.y * resolution.x + pixel.x] = toColorUInt(c);	// convert colour
}

void RenderViewport(std::shared_ptr<PixelBuffer> pixelBuffer, uint32_t frameNumber, float3* accumulationBuffer)
{
	checkCudaErrors(hipGraphicsMapResources(1, &pixelBuffer->GetCudaResource()));
	size_t size = 0;
	uint32_t* devicePtr = 0;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&devicePtr, &size, pixelBuffer->GetCudaResource()));

	uint32_t tx = 16, ty = 16;
	dim3 blocks(pixelBuffer->GetWidth() / tx + 1, pixelBuffer->GetHeight() / ty + 1);
	dim3 threads(tx, ty);

	traceRay<<<blocks, threads>>>(devicePtr, frameNumber, accumulationBuffer);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipGraphicsUnmapResources(1, &pixelBuffer->GetCudaResource(), 0));
}

void SendCameraDataToDevice(Camera* camera)
{
	float3 position = camera->GetPosition();
	float3 forwardDirection = camera->GetForwardDirection();
	float3 rightDirection = camera->GetRightDirection();
	float3 upDirection = cross(rightDirection, forwardDirection);

	float aspectRatio = camera->GetViewportWidth() / (float)camera->GetViewportHeight();
	float halfHeight = camera->GetFocusDist() * tanf(camera->GetVerticalFOV() / 2.0f * M_PI / 180.0f);
	float halfWidth = aspectRatio * halfHeight;

	float3 viewportX = 2 * halfWidth * rightDirection;
	float3 viewportY = 2 * halfHeight * upDirection;
	float3 lowerLeftCorner = position - viewportX / 2.0f - viewportY / 2.0f + forwardDirection * camera->GetFocusDist();

	float lensRadius = camera->GetFocusDist() * tanf(camera->GetDefocusAngle() / 2.0f * M_PI / 180.0f);

	CameraData data = {
		position,
		rightDirection,
		upDirection,
		lensRadius,
		lowerLeftCorner,
		viewportX,
		viewportY,
		make_uint2(camera->GetViewportWidth(), camera->GetViewportHeight())
	};
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cameraData), &data, sizeof(CameraData)));
}

//__device__ float3 getTextureColor(OGLTexture* texture, float u, float v)
//{
//	int texWidth = texture->GetWidth();
//	int texHeight = texture->GetHeight();
//
//	int texX = static_cast<int>(u * texWidth) % texWidth;
//	int texY = static_cast<int>(v * texHeight) % texHeight;
//
//	float3 textureColor = texture->GetPixel(texX, texY);
//
//	return textureColor;
//}