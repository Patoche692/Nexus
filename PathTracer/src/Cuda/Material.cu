#include "Material.cuh"
#include <vector>
#include "../Utils/Utils.h"
#include "Random.cuh"
#include "../Geometry/Ray.h"

__constant__ __device__ Material* materials;


void addMaterialsToDevice(std::vector<Material>& m)
{
	Material* materialsPtr;
	checkCudaErrors(hipGetSymbolAddress((void**)&materialsPtr, materials));
	checkCudaErrors(hipMalloc((void**)&materialsPtr, sizeof(Material) * m.size()));
	checkCudaErrors(hipMemcpy(materialsPtr, &m[0], sizeof(Material) * m.size(), hipMemcpyHostToDevice));
}

void newDeviceMaterial(Material& m, uint32_t size)
{
	Material** materialsSymbolAddress;
	Material* materialsPtr;
	Material* materialsCpy = new Material[size];

	// Retreive the address of materials
	checkCudaErrors(hipGetSymbolAddress((void**)&materialsSymbolAddress, materials));

	// Retrieve the address pointed to by materials
	checkCudaErrors(hipMemcpy(&materialsPtr, materialsSymbolAddress, sizeof(Material*), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(materialsCpy, materialsPtr, sizeof(Material) * (size - 1), hipMemcpyDeviceToHost));
	materialsCpy[size - 1] = m;

	if (size > 1)
		checkCudaErrors(hipFree(materialsPtr));

	Material* temp;
	checkCudaErrors(hipMalloc((void**)&temp, sizeof(Material) * size));

	checkCudaErrors(hipMemcpy(temp, materialsCpy, sizeof(Material) * size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(materials), &temp, sizeof(Material*)));

	delete[] materialsCpy;
}

void changeDeviceMaterial(Material& m, uint32_t id)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(materials), &m, sizeof(Material)));
}


