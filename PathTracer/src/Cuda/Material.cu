#include "Material.cuh"
#include <vector>
#include "../Utils/Utils.h"

__constant__ __device__ CMaterialType* materialTypes;
__constant__ __device__ CMaterial* materials;


void addMaterialsToDevice(std::vector<CMaterial>& m)
{
	CMaterial* materialsPtr;
	checkCudaErrors(hipGetSymbolAddress((void**)&materialsPtr, materials));
	checkCudaErrors(hipMalloc((void**)&materialsPtr, sizeof(CMaterial) * m.size()));
	checkCudaErrors(hipMemcpy(materialsPtr, &m[0], sizeof(CMaterial) * m.size(), hipMemcpyHostToDevice));
}

void addMaterialToDevice(CMaterial& m, CMaterialType mType, uint32_t size)
{
	CMaterial* materialsPtr;
	CMaterial* materialsCpy = new CMaterial[size];

	checkCudaErrors(hipGetSymbolAddress((void**)&materialsPtr, materials));

	checkCudaErrors(hipMemcpyFromSymbol(materialsCpy, HIP_SYMBOL(materials), size - 1));
	materialsCpy[size - 1] = m;

	if (size > 0)
		checkCudaErrors(hipFree(materialsPtr));

	checkCudaErrors(hipMalloc((void**)&materialsPtr, size));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(materials), materialsCpy, size));

	delete[] materialsCpy;

	CMaterialType* materialTypesPtr;
	CMaterialType* materialTypesCpy = new CMaterialType[size];

	checkCudaErrors(hipGetSymbolAddress((void**)&materialTypesPtr, materialTypes));

	checkCudaErrors(hipMemcpyFromSymbol(materialTypesCpy, HIP_SYMBOL(materialTypes), size - 1));
	materialTypesCpy[size - 1] = mType;

	if (size > 0)
		checkCudaErrors(hipFree(materialTypesPtr));

	checkCudaErrors(hipMalloc((void**)&materialTypesPtr, size));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(materialTypes), materialTypesCpy, size));

	delete[] materialTypesCpy;
}
