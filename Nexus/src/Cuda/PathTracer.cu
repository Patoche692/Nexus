#include "hip/hip_runtime.h"
#include "PathTracer.cuh"
#include "Random.cuh"
#include "BSDF/DielectricBSDF.cuh"
#include "BSDF/LambertianBSDF.cuh"
#include "BSDF/BSDF.cuh"
#include "Utils/cuda_math.h"
#include "Utils/Utils.h"
#include "Camera.h"
#include "Geometry/BVH/TLAS.h"
#include "texture_indirect_functions.h"
#include "BSDF/ConductorBSDF.cuh"
#include "BVH/TLAS.cuh"


__constant__ __device__ CameraData cameraData;
__constant__ __device__ SceneData sceneData;
extern __constant__ __device__ Material* materials;
extern __constant__ __device__ hipTextureObject_t* diffuseMaps;
extern __constant__ __device__ hipTextureObject_t* emissiveMaps;
extern __constant__ __device__ TLAS tlas;

inline __device__ uint32_t toColorUInt(float3 color)
{
	float4 clamped = clamp(make_float4(color, 1.0f), make_float4(0.0f), make_float4(1.0f));
	uint8_t red = (uint8_t)(clamped.x * 255.0f);
	uint8_t green = (uint8_t)(clamped.y * 255.0f);
	uint8_t blue = (uint8_t)(clamped.z * 255.0f);
	uint8_t alpha = (uint8_t)(clamped.w * 255.0f);
	 
	return alpha << 24 | blue << 16 | green << 8 | red;
}

// Approximated ACES tonemapping by Krzysztof Narkowicz. See https://graphics-programming.org/resources/tonemapping/index.html
inline __device__ float3 tonemap(float3 color)
{
	// Tungsten renderer filmic tonemapping to compare my results
	//float3 x = fmaxf(make_float3(0.0f), color - 0.004f);
	//return (x * (6.2f * x + 0.5f)) / (x * (6.2f * x + 1.7f) + 0.06f);

	color *= 0.6f; // Exposure
	const float a = 2.51f;
	const float b = 0.03f;
	const float c = 2.43f;
	const float d = 0.59f;
	const float e = 0.14f;
	return clamp((color * (a * color + b)) / (color * (c * color + d) + e), 0.0f, 1.0f);
}

// If necessary, sample the HDR map (from spherical to equirectangular projection)
inline __device__ float3 sampleBackground(float3 direction)
{
	float3 backgroundColor;
	if (sceneData.hasHdrMap)
	{
		// Theta goes from -PI to PI, phi from -PI/2 to PI/2
		const float theta = atan2(direction.z, direction.x);
		const float phi = asin(direction.y);

		// Equirectangular projection
		const float u = (theta + M_PI) * INV_PI * 0.5;
		const float v = 1.0f - (phi + M_PI * 0.5f) * INV_PI;

		backgroundColor = make_float3(tex2D<float4>(sceneData.hdrMap, u, v));
	}
	else
		backgroundColor = make_float3(0.02f);
	return backgroundColor;
}

inline __device__ float3 color(Ray& r, unsigned int& rngState)
{
	Ray currentRay = r;
	float3 currentThroughput = make_float3(1.0f);
	float3 emission = make_float3(0.0f);

	for (int j = 0; j < 15; j++)
	{
		// Reset the hit position and calculate the inverse of the new direction
		currentRay.hit.t = 1e30f;
		currentRay.invDirection = 1 / currentRay.direction;

		IntersectTLAS(tlas, currentRay);

		// If no intersection, sample background
		if (currentRay.hit.t == 1e30f)
		{ 
			float3 backgroundColor = sampleBackground(currentRay.direction);
			return currentThroughput * backgroundColor + emission;
		}

		HitResult hitResult;
		hitResult.p = currentRay.origin + currentRay.direction * currentRay.hit.t;
		hitResult.rIn = currentRay;

		BVHInstance& instance = tlas.blas[currentRay.hit.instanceIdx];
		Triangle& triangle = instance.bvh->triangles[currentRay.hit.triIdx];
		float u = currentRay.hit.u, v = currentRay.hit.v;

		// Interpolating and rotating the normal
		hitResult.normal = u * triangle.normal1 + v * triangle.normal2 + (1 - (u + v)) * triangle.normal0;
		hitResult.normal = normalize(instance.transform.TransformVector(hitResult.normal));

		float3 gNormal = normalize(instance.transform.TransformVector(triangle.Normal()));

		hitResult.material = materials[instance.materialId];

		if (hitResult.material.diffuseMapId == -1)
			hitResult.albedo = hitResult.material.diffuse.albedo;
		else
		{
			float2 uv = u * triangle.texCoord1 + v * triangle.texCoord2 + (1 - (u + v)) * triangle.texCoord0;
			hitResult.material.diffuse.albedo = make_float3(tex2D<float4>(diffuseMaps[hitResult.material.diffuseMapId], uv.x, uv.y));

		}
		if (hitResult.material.emissiveMapId != -1) {
			float2 uv = u * triangle.texCoord1 + v * triangle.texCoord2 + (1 - (u + v)) * triangle.texCoord0;
			hitResult.material.emissive = make_float3(tex2D<float4>(emissiveMaps[hitResult.material.emissiveMapId], uv.x, uv.y));
			//if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0)
			//	printf("emissive map: %f, %f, %f\n", hitResult.material.emissive.x, hitResult.material.emissive.y, hitResult.material.emissive.z);
		}

		// Normal flipping
		//if (dot(hitResult.normal, currentRay.direction) > 0.0f)
		//	hitResult.normal = -hitResult.normal;

		// Invert normals for non transmissive material if the primitive is backfacing the ray
		if (dot(gNormal, currentRay.direction) > 0.0f && (hitResult.material.type != Material::Type::DIELECTRIC || hitResult.material.dielectric.transmittance == 0.0f))
		{
			hitResult.normal = -hitResult.normal;
			gNormal = -gNormal;
		}

		if (fmaxf(hitResult.material.emissive) > 0.0f)
			emission += hitResult.material.emissive * hitResult.material.intensity * currentThroughput;


		// Transform the incoming ray to local space (positive Z axis aligned with shading normal)
		float4 qRotationToZ = getRotationToZAxis(hitResult.normal);
		float3 wi = rotatePoint(qRotationToZ, -hitResult.rIn.direction);

		//bool wiGeometryBackSide = dot(wi, gNormal) < 0.0f;
		//bool wiShadingBackSide = dot(wi, hitResult.normal) < 0.0f;

		//if (wiGeometryBackSide != wiShadingBackSide)
		//	continue;

		float3 throughput;
		float3 wo;

		bool scattered = false;
		switch (hitResult.material.type)
		{
		case Material::Type::DIFFUSE:
			scattered = BSDF::Sample<LambertianBSDF>(hitResult, wi, wo, throughput, rngState);
			break;
		case Material::Type::DIELECTRIC:
			scattered = BSDF::Sample<DielectricBSDF>(hitResult, wi, wo, throughput, rngState);
			break;
		case Material::Type::CONDUCTOR:
			scattered = BSDF::Sample<ConductorBSDF>(hitResult, wi, wo, throughput, rngState);
			break;
		default:
			break;
		}

		if (scattered)
		{
			// Inverse ray transformation to world space
			wo = normalize(rotatePoint(invertRotation(qRotationToZ), wo));
			bool woGeometryBackSide = dot(wo, gNormal) < 0.0f;
			bool woShadingBackSide = dot(wo, hitResult.normal) < 0.0f;

			if (woGeometryBackSide == woShadingBackSide)
			{
				currentThroughput *= throughput;
				float offsetDirection = woGeometryBackSide ? -1.0f : 1.0f;
				currentRay.origin = hitResult.p + offsetDirection * 1.0e-4 * hitResult.normal;
				currentRay.direction = wo;
			}
		}

		// Russian roulette
		if (j > 2)
		{
			float p = clamp(fmax(currentThroughput.x, fmax(currentThroughput.y, currentThroughput.z)), 0.01f, 1.0f);
			if (Random::Rand(rngState) < p)
			{
				// To get unbiased results, we need to increase the contribution of
				// the non-terminated rays with their probability of being terminated
				currentThroughput *= 1.0f / p;
			}
			else
				return emission;
		}
	}

	return emission;
}

__global__ void traceRay(uint32_t* outBufferPtr, uint32_t frameNumber, float3* accumulationBuffer)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	uint2 pixel = make_uint2(i, j);

	uint2 resolution = cameraData.resolution;

	if (pixel.x >= resolution.x || pixel.y >= resolution.y)
		return;

	unsigned int rngState = Random::InitRNG(pixel, resolution, frameNumber);

	// Avoid using modulo, it significantly impacts performance
	float x = (pixel.x + Random::Rand(rngState)) / (float)resolution.x;
	float y = (pixel.y + Random::Rand(rngState)) / (float)resolution.y;

	float2 rd = cameraData.lensRadius * Random::RandomInUnitDisk(rngState);
	float3 offset = cameraData.right * rd.x + cameraData.up * rd.y;

	Ray ray(
		//make_float3(2, 0, 0), make_float3(-1, 0.001, 0.001)
		cameraData.position + offset,
		normalize(cameraData.lowerLeftCorner + x * cameraData.viewportX + y * cameraData.viewportY - cameraData.position - offset)
	);

	float3 c = color(ray, rngState);
	if (frameNumber == 1)
		accumulationBuffer[pixel.y * resolution.x + pixel.x] = c;
	else
		accumulationBuffer[pixel.y * resolution.x + pixel.x] += c;

	c = accumulationBuffer[pixel.y * resolution.x + pixel.x] / frameNumber;

	outBufferPtr[pixel.y * resolution.x + pixel.x] = toColorUInt(Utils::LinearToGamma(tonemap(c)));
}

void RenderViewport(std::shared_ptr<PixelBuffer> pixelBuffer, uint32_t frameNumber, float3* accumulationBuffer)
{
	checkCudaErrors(hipGraphicsMapResources(1, &pixelBuffer->GetCudaResource()));
	size_t size = 0;
	uint32_t* devicePtr = 0;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&devicePtr, &size, pixelBuffer->GetCudaResource()));

	uint32_t tx = 16, ty = 16;
	dim3 blocks(pixelBuffer->GetWidth() / tx + 1, pixelBuffer->GetHeight() / ty + 1);
	dim3 threads(tx, ty);

	traceRay<<<blocks, threads>>>(devicePtr, frameNumber, accumulationBuffer);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipGraphicsUnmapResources(1, &pixelBuffer->GetCudaResource(), 0));
}

void InitDeviceSceneData()
{
	SceneData scene;
	scene.hasHdrMap = false;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sceneData), &scene, sizeof(SceneData)));
}

void SendHDRMapToDevice(const Texture& map)
{
	// Channel descriptor for 4 Channels (RGBA)
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipArray_t cuArray;
	checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, map.width, map.height));

	const size_t spitch = map.width * 4 * sizeof(float);
	checkCudaErrors(hipMemcpy2DToArray(cuArray, 0, 0, map.pixels, spitch, map.width * 4 * sizeof(float), map.height, hipMemcpyHostToDevice));

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	hipTextureObject_t texObject = 0;
	checkCudaErrors(hipCreateTextureObject(&texObject, &resDesc, &texDesc, NULL));

	SceneData scene;
	scene.hasHdrMap = true;
	scene.hdrMap = texObject;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sceneData), &scene, sizeof(SceneData)));
}

void SendCameraDataToDevice(Camera* camera)
{
	float3 position = camera->GetPosition();
	float3 forwardDirection = camera->GetForwardDirection();
	float3 rightDirection = camera->GetRightDirection();
	float3 upDirection = cross(rightDirection, forwardDirection);

	float aspectRatio = camera->GetViewportWidth() / (float)camera->GetViewportHeight();
	float halfHeight = camera->GetFocusDist() * tanf(camera->GetVerticalFOV() / 2.0f * M_PI / 180.0f);
	float halfWidth = aspectRatio * halfHeight;

	float3 viewportX = 2 * halfWidth * rightDirection;
	float3 viewportY = 2 * halfHeight * upDirection;
	float3 lowerLeftCorner = position - viewportX / 2.0f - viewportY / 2.0f + forwardDirection * camera->GetFocusDist();

	float lensRadius = camera->GetFocusDist() * tanf(camera->GetDefocusAngle() / 2.0f * M_PI / 180.0f);

	CameraData data = {
		position,
		rightDirection,
		upDirection,
		lensRadius,
		lowerLeftCorner,
		viewportX,
		viewportY,
		make_uint2(camera->GetViewportWidth(), camera->GetViewportHeight())
	};
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cameraData), &data, sizeof(CameraData)));
}