#include "hip/hip_runtime.h"
#include "AssetManager.cuh"
#include <vector>
#include <map>
#include "CudaMemory.cuh"
#include "../Utils/Utils.h"
#include "Random.cuh"
#include "Geometry/Ray.h"
#include "Geometry/Triangle.h"

//__constant__ __device__ Material* materials;
//__constant__ __device__ hipTextureObject_t* diffuseMaps;
//__constant__ __device__ hipTextureObject_t* emissiveMaps;
//__constant__ __device__ BVH8* bvhs;
//__constant__ __device__ TLAS tlas;
//
//
//void newDeviceMesh(Mesh& mesh, uint32_t size)
//{
//	//Mesh** meshesSymbolAddress;
//
//	//// Retreive the address of meshes
//	//checkCudaErrors(hipGetSymbolAddress((void**)&meshesSymbolAddress, bvhs));
//
//	//Triangle* triangles = CudaMemory::Allocate<Triangle>(mesh.nTriangles);
//	//CudaMemory::MemCpy(triangles, mesh.triangles, mesh.nTriangles, hipMemcpyHostToDevice);
//
//	//Mesh newMesh = mesh;
//	//newMesh.triangles = triangles;
//
//	//CudaMemory::ResizeDeviceArray(meshesSymbolAddress, size);
//
//	//CudaMemory::SetToIndex(meshesSymbolAddress, size - 1, newMesh);
//}
//
//void newDeviceMaterial(Material& material, uint32_t size)
//{
//	Material** materialsSymbolAddress;
//
//	// Retreive the address of materials
//	checkCudaErrors(hipGetSymbolAddress((void**)&materialsSymbolAddress, materials));
//
//	CudaMemory::ResizeDeviceArray(materialsSymbolAddress, size);
//
//	CudaMemory::SetToIndex(materialsSymbolAddress, size - 1, material);
//}
//
//void newDeviceTexture(Texture& texture, uint32_t size) {
//	
//	hipTextureObject_t** texturesSymbolAddress;
//
//	if (texture.type == Texture::Type::DIFFUSE) {
//		checkCudaErrors(hipGetSymbolAddress((void**)&texturesSymbolAddress, diffuseMaps));
//	}
//	else if (texture.type == Texture::Type::EMISSIVE) {
//		checkCudaErrors(hipGetSymbolAddress((void**)&texturesSymbolAddress, emissiveMaps));
//	}
//		
//
//	CudaMemory::ResizeDeviceArray(texturesSymbolAddress, size);
//
//	// Channel descriptor for 4 Channels (RGBA)
//	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
//	hipArray_t cuArray;
//	checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, texture.width, texture.height));
//
//	const size_t spitch = texture.width * 4 * sizeof(float);
//	checkCudaErrors(hipMemcpy2DToArray(cuArray, 0, 0, texture.pixels, spitch, texture.width * 4 * sizeof(float), texture.height, hipMemcpyHostToDevice));
//
//	hipResourceDesc resDesc;
//	memset(&resDesc, 0, sizeof(resDesc));
//	resDesc.resType = hipResourceTypeArray;
//	resDesc.res.array.array = cuArray;
//
//	hipTextureDesc texDesc;
//	memset(&texDesc, 0, sizeof(texDesc));
//	texDesc.addressMode[0] = hipAddressModeWrap;
//	texDesc.addressMode[1] = hipAddressModeWrap;
//	texDesc.filterMode = hipFilterModeLinear;
//	texDesc.readMode = hipReadModeElementType;
//	texDesc.normalizedCoords = 1;
//
//	hipTextureObject_t texObject = 0;
//	checkCudaErrors(hipCreateTextureObject(&texObject, &resDesc, &texDesc, NULL));
//
//	CudaMemory::SetToIndex(texturesSymbolAddress, size - 1, texObject);
//}
//
//__global__ void freeMeshesKernel(int meshesCount)
//{
//	for (int i = 0; i < meshesCount; i++)
//	{
//		free(bvhs[i].triangles);
//	}
//	free(bvhs);
//}
//
//void freeDeviceMeshes(int meshesCount)
//{
//	freeMeshesKernel<<<1, 1>>>(meshesCount);
//}
//
//__global__ void freeMaterialsKernel()
//{
//	free(materials);
//}
//
//void freeDeviceMaterials()
//{
//	freeMaterialsKernel<<<1, 1>>>();
//}
//
//__global__ void freeTexturesKernel(int texturesCount)
//{
//	//for (int i = 0; i < texturesCount; i++)
//	//	free(textures[i].pixels);
//
//	//free(textures);
//}
//
//void freeDeviceTextures(int texturesCount)
//{
//	freeTexturesKernel<<<1, 1>>>(texturesCount);
//}
//
//void cpyMaterialToDevice(Material& m, uint32_t id)
//{
//	Material** materialsSymbolAddress;
//
//	// Retreive the address of materials
//	checkCudaErrors(hipGetSymbolAddress((void**)&materialsSymbolAddress, materials));
//
//	CudaMemory::SetToIndex(materialsSymbolAddress, id, m);
//}
//
//BVH2* newDeviceBVH(BVH2& bvh)
//{
//	Triangle* triangles = CudaMemory::Allocate<Triangle>(bvh.triCount);
//	BVH2Node* nodes = CudaMemory::Allocate<BVH2Node>(bvh.triCount * 2);
//	uint32_t* triangleIdx = CudaMemory::Allocate<uint32_t>(bvh.triCount);
//
//	CudaMemory::MemCpy(triangles, bvh.triangles, bvh.triCount, hipMemcpyHostToDevice);
//	CudaMemory::MemCpy(nodes, bvh.nodes, bvh.triCount * 2, hipMemcpyHostToDevice);
//	CudaMemory::MemCpy(triangleIdx, bvh.triangleIdx, bvh.triCount, hipMemcpyHostToDevice);
//
//	BVH2 newBvh;
//	newBvh.triangles = triangles;
//	newBvh.nodes = nodes;
//	newBvh.triangleIdx = triangleIdx;
//	newBvh.triCount = bvh.triCount;
//
//	newBvh.nodesUsed = bvh.nodesUsed;
//
//	BVH2* bvhPtr = CudaMemory::Allocate<BVH2>(1);
//	CudaMemory::MemCpy(bvhPtr, &newBvh, 1, hipMemcpyHostToDevice);
//
//	// TODO: Move all structures to the GPU. For now, avoid calling delete on a device ptr
//	newBvh.triangles = nullptr;
//	newBvh.nodes = nullptr;
//	newBvh.triangleIdx = nullptr;
//
//	return bvhPtr;
//}
//
//BVH8* newDeviceBVH8(BVH8& bvh)
//{
//	Triangle* triangles = CudaMemory::Allocate<Triangle>(bvh.triCount);
//	BVH8Node* nodes = CudaMemory::Allocate<BVH8Node>(bvh.triCount * 2);
//	uint32_t* triangleIdx = CudaMemory::Allocate<uint32_t>(bvh.triCount);
//
//	CudaMemory::MemCpy(triangles, bvh.triangles, bvh.triCount, hipMemcpyHostToDevice);
//	CudaMemory::MemCpy(nodes, bvh.nodes, bvh.triCount * 2, hipMemcpyHostToDevice);
//	CudaMemory::MemCpy(triangleIdx, bvh.triangleIdx, bvh.triCount, hipMemcpyHostToDevice);
//
//	BVH8 newBvh;
//	newBvh.triangles = triangles;
//	newBvh.nodes = nodes;
//	newBvh.triangleIdx = triangleIdx;
//	newBvh.triCount = bvh.triCount;
//
//	newBvh.nodesUsed = bvh.nodesUsed;
//
//	BVH8* bvhPtr = CudaMemory::Allocate<BVH8>(1);
//	CudaMemory::MemCpy(bvhPtr, &newBvh, 1, hipMemcpyHostToDevice);
//
//	// TODO: Move all structures to the GPU. For now, avoid calling delete on a device ptr
//	newBvh.triangles = nullptr;
//	newBvh.nodes = nullptr;
//	newBvh.triangleIdx = nullptr;
//
//	return bvhPtr;
//}
//
//void newDeviceTLAS(TLAS& tl)
//{
//	TLASNode* tlasNodes = CudaMemory::Allocate<TLASNode>(tl.blasCount * 2);
//	uint32_t *nodesIdx = CudaMemory::Allocate<uint32_t>(tl.blasCount);
//	BVHInstance* instances = CudaMemory::Allocate<BVHInstance>(tl.blasCount);
//
//	CudaMemory::MemCpy(tlasNodes, tl.nodes, tl.blasCount * 2, hipMemcpyHostToDevice);
//	CudaMemory::MemCpy(nodesIdx, tl.nodesIdx, tl.blasCount, hipMemcpyHostToDevice);
//
//	// Map from cpu memory to device memory
//	std::map<BVH8*, BVH8*> bvhMap;
//	for (int i = 0; i < tl.blasCount; i++)
//	{
//		if (!bvhMap.count(tl.blas[i].bvh))
//		{
//			bvhMap[tl.blas[i].bvh] = newDeviceBVH8(*tl.blas[i].bvh);
//		}
//		BVHInstance instance = tl.blas[i];
//		instance.bvh = bvhMap[tl.blas[i].bvh];
//		CudaMemory::MemCpy(instances + i, &instance, 1, hipMemcpyHostToDevice);
//	}
//
//	TLAS newTlas;
//	newTlas.blasCount = tl.blasCount;
//	newTlas.nodesUsed = tl.nodesUsed;
//	newTlas.nodes = tlasNodes;
//	newTlas.blas = instances;
//	newTlas.nodesIdx = nodesIdx;
//
//	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(tlas), &newTlas, sizeof(TLAS)));
//}
//
//void updateDeviceTLAS(TLAS& tl)
//{
//	TLAS tlasCpy;
//	BVHInstance* instancesCpy = new BVHInstance[tl.blasCount];
//	checkCudaErrors(hipMemcpyFromSymbol(&tlasCpy, HIP_SYMBOL(tlas), sizeof(TLAS)));
//
//	// Update the nodes
//	CudaMemory::MemCpy(tlasCpy.nodes, tl.nodes, tl.blasCount * 2, hipMemcpyHostToDevice);
//
//	// TODO: handle the case when tl.blasCount has changed (new instance or deleted instance)
//	CudaMemory::MemCpy(instancesCpy, tlasCpy.blas, tl.blasCount, hipMemcpyDeviceToHost);
//
//	for (int i = 0; i < tl.blasCount; i++)
//	{
//		BVH8* bvhBackup = instancesCpy[i].bvh;
//		instancesCpy[i].bvh = tl.blas[i].bvh;
//		instancesCpy[i].SetTransform(tl.blas[i].transform);
//		instancesCpy[i].materialId = tl.blas[i].materialId;
//		instancesCpy[i].bvh = bvhBackup;
//	}
//	
//	// Copy the instances back to the GPU
//	CudaMemory::MemCpy(tlasCpy.blas, instancesCpy, tl.blasCount, hipMemcpyHostToDevice);
//
//	delete[] instancesCpy;
//}
//
//__global__ void freeDeviceTLASKernel()
//{
//	for (int i = 0; i < tlas.blasCount; i++)
//	{
//		BVH8* bvh = tlas.blas[i].bvh;
//		free(bvh->nodes);
//		free(bvh->triangles);
//		free(bvh->triangleIdx);
//		free(bvh);
//	}
//	free(tlas.blas);
//	free(tlas.nodes);
//	free(tlas.nodesIdx);
//}
//
//void freeDeviceTLAS()
//{
//	freeDeviceTLASKernel<<<1, 1>>>();
//}
//
//Material** getMaterialSymbolAddress()
//{
//	Material** materialSymbolAddress;
//	checkCudaErrors(hipGetSymbolAddress((void**)&materialSymbolAddress, materials));
//	return materialSymbolAddress;
//}
//
//Mesh** getMeshSymbolAddress()
//{
//	Mesh** meshSymbolAddress;
//	checkCudaErrors(hipGetSymbolAddress((void**)&meshSymbolAddress, bvhs));
//	return meshSymbolAddress;
//}
//
//
