#include "hip/hip_runtime.h"
#include "PathTracer.cuh"
#include "Cuda/Random.cuh"
#include "Cuda/BSDF/LambertianBSDF.cuh"
#include "Cuda/BSDF/DielectricBSDF.cuh"
#include "Cuda/BSDF/PlasticBSDF.cuh"
#include "Cuda/BSDF/ConductorBSDF.cuh"
#include "Cuda/BSDF/BSDF.cuh"
#include "Utils/cuda_math.h"
#include "Utils/Utils.h"
#include "texture_indirect_functions.h"
#include "Cuda/BVH/BVH8Traversal.cuh"
#include "Cuda/Scene/Scene.cuh"
#include "Cuda/Scene/Camera.cuh"
#include "Cuda/Sampler.cuh"


__device__ __constant__ uint32_t frameNumber;
__device__ __constant__ uint32_t bounce;

__device__ __constant__ float3* accumulationBuffer;
__device__ __constant__ uint32_t* renderBuffer;

__device__ __constant__ D_Scene scene;
__device__ __constant__ D_PathStateSAO pathState;
__device__ __constant__ D_TraceRequestSAO traceRequest;
__device__ __constant__ D_ShadowTraceRequestSAO shadowTraceRequest;

__device__ __constant__ D_MaterialRequestSAO diffuseMaterialBuffer;
__device__ __constant__ D_MaterialRequestSAO plasticMaterialBuffer;
__device__ __constant__ D_MaterialRequestSAO dielectricMaterialBuffer;
__device__ __constant__ D_MaterialRequestSAO conductorMaterialBuffer;

__device__ D_PixelQuery pixelQuery;
__device__ D_QueueSize queueSize;


inline __device__ uint32_t ToColorUInt(float3 color)
{
	float4 clamped = clamp(make_float4(color, 1.0f), make_float4(0.0f), make_float4(1.0f));
	uint8_t red = (uint8_t)(clamped.x * 255.0f);
	uint8_t green = (uint8_t)(clamped.y * 255.0f);
	uint8_t blue = (uint8_t)(clamped.z * 255.0f);
	uint8_t alpha = (uint8_t)(clamped.w * 255.0f);
	 
	return alpha << 24 | blue << 16 | green << 8 | red;
}

// Approximated ACES tonemapping by Krzysztof Narkowicz. See https://graphics-programming.org/resources/tonemapping/index.html
inline __device__ float3 Tonemap(float3 color)
{
	// Tungsten renderer filmic tonemapping to compare my results
	//float3 x = fmaxf(make_float3(0.0f), color - 0.004f);
	//return (x * (6.2f * x + 0.5f)) / (x * (6.2f * x + 1.7f) + 0.06f);

	color *= 0.6f; // Exposure
	const float a = 2.51f;
	const float b = 0.03f;
	const float c = 2.43f;
	const float d = 0.59f;
	const float e = 0.14f;
	return clamp((color * (a * color + b)) / (color * (c * color + d) + e), 0.0f, 1.0f);
}

// If necessary, sample the HDR map (from spherical to equirectangular projection)
inline __device__ float3 SampleBackground(const D_Scene& scene, float3 direction)
{
	float3 backgroundColor;
	if (scene.hasHdrMap)
	{
		// Theta goes from -PI to PI, phi from -PI/2 to PI/2
		const float theta = atan2(direction.z, direction.x);
		const float phi = asin(direction.y);

		// Equirectangular projection
		const float u = (theta + PI) * INV_PI * 0.5;
		const float v = 1.0f - (phi + PI * 0.5f) * INV_PI;

		backgroundColor = make_float3(tex2D<float4>(scene.hdrMap, u, v));
	}
	else
		backgroundColor = scene.renderSettings.backgroundColor * scene.renderSettings.backgroundIntensity;
	return backgroundColor;
}

__global__ void GenerateKernel()
{
	const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;

	const D_Camera camera = scene.camera;
	uint2 resolution = camera.resolution;

	if (index >= resolution.x * resolution.y)
		return;

	const uint32_t j = index / resolution.x;
	const uint32_t i = index - j * resolution.x;

	const uint2 pixel = make_uint2(i, j);

	unsigned int rngState = Random::InitRNG(pixel, resolution, frameNumber);

	// Normalized jittered coordinates
	const float x = (pixel.x + Random::Rand(rngState)) / (float)resolution.x;
	const float y = (pixel.y + Random::Rand(rngState)) / (float)resolution.y;

	float2 rd = camera.lensRadius * Random::RandomInUnitDisk(rngState);
	float3 offset = camera.right * rd.x + camera.up * rd.y;

	D_Ray ray(
		camera.position + offset,
		normalize(camera.lowerLeftCorner + x * camera.viewportX + y * camera.viewportY - camera.position - offset)
	);

	if (index == 0)
		queueSize.traceSize[0] = resolution.x * resolution.y;

	traceRequest.ray.origin[index] = ray.origin;
	traceRequest.ray.direction[index] = ray.direction;
	traceRequest.pixelIdx[index] = index;
}


__global__ void TraceKernel()
{
	BVH8Trace(traceRequest, queueSize.traceSize[bounce], &queueSize.traceCount[bounce]);
}

__global__ void TraceShadowKernel()
{
	BVH8TraceShadow(shadowTraceRequest, queueSize.traceShadowSize[bounce], &queueSize.traceShadowCount[bounce], pathState.radiance);
}


__global__ void LogicKernel()
{
	const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= queueSize.traceSize[bounce - 1])
		return;

	uint32_t rngState = Random::InitRNG(index, scene.camera.resolution, frameNumber);

	const D_Intersection intersection = traceRequest.intersection.Get(index);
	const D_Ray ray(traceRequest.ray.origin[index], traceRequest.ray.direction[index]);
	const uint32_t pixelIdx = traceRequest.pixelIdx[index];

	const float3 throughput = bounce == 1 ? make_float3(1.0f) : pathState.throughput[pixelIdx];

	// If no intersection, sample background
	if (intersection.hitDistance == 1e30f)
	{
		float3 backgroundColor = SampleBackground(scene, ray.direction);
		if (bounce == 1)
			pathState.radiance[pixelIdx] = throughput * backgroundColor;
		else
			pathState.radiance[pixelIdx] += throughput * backgroundColor;

		if (bounce == 1 && pixelQuery.pixelIdx == pixelIdx)
			pixelQuery.instanceIdx = -1;

		return;
	}

	if (bounce == 1 && pixelQuery.pixelIdx == pixelIdx)
		pixelQuery.instanceIdx = intersection.instanceIdx;

	const D_BVHInstance instance = blas[intersection.instanceIdx];
	D_Material material = scene.materials[instance.materialId];

	const D_Triangle triangle = bvhs[instance.bvhIdx].triangles[intersection.triIdx];
	float u = intersection.u, v = intersection.v;

	if (material.emissiveMapId != -1)
	{
		float2 uv = u * triangle.texCoord1 + v * triangle.texCoord2 + (1 - (u + v)) * triangle.texCoord0;
		material.emissive = make_float3(tex2D<float4>(scene.emissiveMaps[material.emissiveMapId], uv.x, uv.y));
	}

	const float3 edge1 = triangle.pos1 - triangle.pos0;
	const float3 edge2 = triangle.pos2 - triangle.pos0;
	float3 p = triangle.pos0 + intersection.u * edge1 + intersection.v * edge2;
	p = instance.transform.TransformPoint(p);

	// Interpolating and rotating the normal
	float3 normal = u * triangle.normal1 + v * triangle.normal2 + (1 - (u + v)) * triangle.normal0;
	normal = normalize(instance.transform.TransformVector(normal));

	float3 radiance = make_float3(0.0f);

	if (fmaxf(material.emissive * material.intensity) > 0.0f)
	{
		float weight = 1.0f;

		// Not using MIS for primary rays
		if (scene.renderSettings.useMIS && bounce > 1)
		{
			const float lastPdf = pathState.lastPdf[pixelIdx];

			const float cosThetaO = fabs(dot(normal, ray.direction));

			const float dSquared = Square(intersection.hitDistance);

			float lightPdf = 1.0f / (scene.lightCount * bvhs[instance.bvhIdx].triCount * triangle.Area());
			// Transform pdf over an area to pdf over directions
			lightPdf *= dSquared / cosThetaO;

			weight = Sampler::PowerHeuristic(lastPdf, lightPdf);
		}
		radiance = weight * material.emissive * material.intensity * throughput;

	}

	if (bounce == 1)
		pathState.radiance[pixelIdx] = radiance;
	else
		pathState.radiance[pixelIdx] += radiance;

	if (bounce == scene.renderSettings.pathLength)
		return;

	// Russian roulette
	float probability = fmaxf(throughput);// clamp(fmaxf(currentThroughput), 0.01f, 1.0f);
	if (Random::Rand(rngState) < probability)
	{
		// To get unbiased results, we need to increase the contribution of
		// the non-terminated rays with their probability of being terminated
		pathState.throughput[pixelIdx] = throughput / probability;
	}
	else
		return;

	int32_t requestIdx;
	switch (material.type)
	{
	case D_Material::D_Type::DIFFUSE:
		requestIdx = atomicAdd(&queueSize.diffuseSize[bounce], 1);
		diffuseMaterialBuffer.intersection.Set(requestIdx, intersection);
		diffuseMaterialBuffer.rayDirection[requestIdx] = ray.direction;
		diffuseMaterialBuffer.pixelIdx[requestIdx] = pixelIdx;
		break;
	case D_Material::D_Type::PLASTIC:
		requestIdx = atomicAdd(&queueSize.plasticSize[bounce], 1);
		plasticMaterialBuffer.intersection.Set(requestIdx, intersection);
		plasticMaterialBuffer.rayDirection[requestIdx] = ray.direction;
		plasticMaterialBuffer.pixelIdx[requestIdx] = pixelIdx;
		break;
	case D_Material::D_Type::DIELECTRIC:
		requestIdx = atomicAdd(&queueSize.dielectricSize[bounce], 1);
		dielectricMaterialBuffer.intersection.Set(requestIdx, intersection);
		dielectricMaterialBuffer.rayDirection[requestIdx] = ray.direction;
		dielectricMaterialBuffer.pixelIdx[requestIdx] = pixelIdx;
		break;
	case D_Material::D_Type::CONDUCTOR:
		requestIdx = atomicAdd(&queueSize.conductorSize[bounce], 1);
		conductorMaterialBuffer.intersection.Set(requestIdx, intersection);
		conductorMaterialBuffer.rayDirection[requestIdx] = ray.direction;
		conductorMaterialBuffer.pixelIdx[requestIdx] = pixelIdx;
		break;
	default:
		break;
	}
}


template<typename BSDF>
inline __device__ void NextEventEstimation(
	const float3 wi,
	const D_Material& material,
	const D_Intersection& intersection,
	const float3 hitPoint,
	const float3 normal,
	const float3 hitGNormal,
	const float3 throughput,
	const uint32_t pixelIdx,
	unsigned int& rngState
) {
	D_Light light = Sampler::UniformSampleLights(scene.lights, scene.lightCount, rngState);

	if (light.type == D_Light::Type::MESH_LIGHT)
	{
		D_BVHInstance instance = blas[light.mesh.meshId];

		uint32_t triangleIdx;
		float2 uv;
		Sampler::UniformSampleMesh(bvhs[instance.bvhIdx], rngState, triangleIdx, uv);

		D_Triangle triangle = bvhs[instance.bvhIdx].triangles[triangleIdx];

		const float3 edge1 = triangle.pos1 - triangle.pos0;
		const float3 edge2 = triangle.pos2 - triangle.pos0;
		float3 p = triangle.pos0 + uv.x * edge1 + uv.y * edge2;
		p = instance.transform.TransformPoint(p);

		const float3 lightGNormal = normalize(instance.transform.TransformVector(triangle.Normal()));

		float3 lightNormal = uv.x * triangle.normal1 + uv.y * triangle.normal2 + (1 - (uv.x + uv.y)) * triangle.normal0;
		lightNormal = normalize(instance.transform.TransformVector(lightNormal));

		// TODO: change
		//bool woShadingBackSide = wo.z < 0.0f;
		//bool woGeometryBackSide = dot(-hitResult.rIn.direction, hitGNormal) < 0.0f;

		//if (wiGeometryBackSide != wiShadingBackSide)
		//	return make_float3(0.0f);

		D_Ray shadowRay;
		float offsetDirection = 1.0;// wiGeometryBackSide ? -1.0f : 1.0f;
		shadowRay.origin = hitPoint + offsetDirection * 1.0e-4f * normal;

		const float3 toLight = p - shadowRay.origin;

		const float distance = length(toLight);
		shadowRay.direction = toLight / distance;
		shadowRay.invDirection = 1.0f / shadowRay.direction;

		float4 qRotationToZ = getRotationToZAxis(normal);
		const float3 wo = rotatePoint(qRotationToZ, shadowRay.direction);

		const float cosThetaO = fabs(dot(lightNormal, shadowRay.direction));

		const float dSquared = dot(toLight, toLight);

		float lightPdf = 1.0f / (scene.lightCount * bvhs[instance.bvhIdx].triCount * triangle.Area());
		// Transform pdf over an area to pdf over directions
		lightPdf *= dSquared / cosThetaO;

		if (!Sampler::IsPdfValid(lightPdf))
			return;

		const D_Material lightMaterial = scene.materials[instance.materialId];

		float3 sampleThroughput;
		float bsdfPdf;

		bool sampleIsValid = D_BSDF::Eval<BSDF>(material, wi, wo, sampleThroughput, bsdfPdf);

		if (!sampleIsValid)
			return;

		//const float weight = 1.0f;
		const float weight = Sampler::PowerHeuristic(lightPdf, bsdfPdf);

		float3 emissive;
		if (lightMaterial.emissiveMapId != -1)
		{
			float2 texUv = uv.x * triangle.texCoord1 + uv.y * triangle.texCoord2 + (1 - (uv.x + uv.y)) * triangle.texCoord0;
			emissive = make_float3(tex2D<float4>(scene.emissiveMaps[lightMaterial.emissiveMapId], texUv.x, texUv.y));
		}
		else
			emissive = lightMaterial.emissive;

		const float3 radiance = weight * throughput * sampleThroughput * emissive * lightMaterial.intensity / lightPdf;

		const int32_t index = atomicAdd(&queueSize.traceShadowSize[bounce], 1);
		shadowTraceRequest.hitDistance[index] = distance;
		shadowTraceRequest.radiance[index] = radiance;
		shadowTraceRequest.ray.Set(index, shadowRay);
		shadowTraceRequest.pixelIdx[index] = pixelIdx;
	}
}


template<typename BSDF>
inline __device__ void Shade(D_MaterialRequestSAO materialRequest, int32_t size)
{
	const int32_t requestIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (requestIdx >= size)
		return;

	const D_Intersection intersection = materialRequest.intersection.Get(requestIdx);
	const float3 rayDirection = materialRequest.rayDirection[requestIdx];
	const uint32_t pixelIdx = materialRequest.pixelIdx[requestIdx];

	float3 throughput = bounce == 1 ? make_float3(1.0f) : pathState.throughput[pixelIdx];

	uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t rngState = Random::InitRNG(index, scene.camera.resolution, frameNumber);

	const D_BVHInstance instance = blas[intersection.instanceIdx];
	const D_Triangle triangle = bvhs[instance.bvhIdx].triangles[intersection.triIdx];

	D_Material material = scene.materials[instance.materialId];

	const float u = intersection.u, v = intersection.v;

	const float3 edge1 = triangle.pos1 - triangle.pos0;
	const float3 edge2 = triangle.pos2 - triangle.pos0;
	float3 p = triangle.pos0 + intersection.u * edge1 + intersection.v * edge2;
	p = instance.transform.TransformPoint(p);

	float3 normal = u * triangle.normal1 + v * triangle.normal2 + (1 - (u + v)) * triangle.normal0;
	normal = normalize(instance.transform.TransformVector(normal));

	float3 gNormal = normalize(instance.transform.TransformVector(triangle.Normal()));

	if (material.diffuseMapId != -1)
	{
		float2 uv = u * triangle.texCoord1 + v * triangle.texCoord2 + (1 - (u + v)) * triangle.texCoord0;
		material.diffuse.albedo = make_float3(tex2D<float4>(scene.diffuseMaps[material.diffuseMapId], uv.x, uv.y));
	}

	// Invert normals for non transmissive material if the primitive is backfacing the ray
	if (dot(gNormal, rayDirection) > 0.0f && (material.type != D_Material::D_Type::DIELECTRIC))
	{
		normal = -normal;
		gNormal = -gNormal;
	}

	float4 qRotationToZ = getRotationToZAxis(normal);
	float3 wi = rotatePoint(qRotationToZ, -rayDirection);

	if (scene.renderSettings.useMIS)
		NextEventEstimation<BSDF>(wi, material, intersection, p, normal, gNormal, throughput, pixelIdx, rngState);

	float3 wo, sampleThroughput;
	float pdf;

	bool scattered = D_BSDF::Sample<BSDF>(material, wi, wo, sampleThroughput, pdf, rngState);

	if (!scattered)
		return;

	throughput *= sampleThroughput;

	wo = normalize(rotatePoint(invertRotation(qRotationToZ), wo));
	bool woGeometryBackSide = dot(wo, gNormal) < 0.0f;
	bool woShadingBackSide = dot(wo, normal) < 0.0f;

	// If sample is valid, write trace request in the path state
	if (woGeometryBackSide == woShadingBackSide)
	{
		float offsetDirection = woGeometryBackSide ? -1.0f : 1.0f;
		const D_Ray scatteredRay(p + offsetDirection * 1.0e-4 * normal, wo);

		const int32_t traceRequestIdx = atomicAdd(&queueSize.traceSize[bounce], 1);
		traceRequest.ray.Set(traceRequestIdx, scatteredRay);
		traceRequest.pixelIdx[traceRequestIdx] = pixelIdx;

		pathState.throughput[pixelIdx] = throughput;


		pathState.lastPdf[pixelIdx] = pdf;
	}
}

__global__ void DiffuseMaterialKernel()
{
	Shade<D_LambertianBSDF>(diffuseMaterialBuffer, queueSize.diffuseSize[bounce]);
}

__global__ void PlasticMaterialKernel()
{
	Shade<D_PlasticBSDF>(plasticMaterialBuffer, queueSize.plasticSize[bounce]);
}

__global__ void DielectricMaterialKernel()
{
	Shade<D_DielectricBSDF>(dielectricMaterialBuffer, queueSize.dielectricSize[bounce]);
}

__global__ void ConductorMaterialKernel()
{
	//Shade<D_ConductorBSDF>(conductorMaterialBuffer, queueSize.conductorSize[bounce]);
}

__global__ void AccumulateKernel()
{
	const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;

	const uint2 resolution = scene.camera.resolution;

	if (index >= resolution.x * resolution.y)
		return;

	if (frameNumber == 1)
		accumulationBuffer[index] = pathState.radiance[index];
	else
		accumulationBuffer[index] += pathState.radiance[index];

	float3 c = accumulationBuffer[index] / frameNumber;

	renderBuffer[index] = ToColorUInt(Utils::LinearToGamma(Tonemap(c)));
}

D_Scene* GetDeviceSceneAddress()
{
	D_Scene* deviceScene;
	CheckCudaErrors(hipGetSymbolAddress((void**)&deviceScene, scene));
	return deviceScene;
}

float3** GetDeviceAccumulationBufferAddress()
{
	float3** buffer;
	CheckCudaErrors(hipGetSymbolAddress((void**)&buffer, accumulationBuffer));
	return buffer;
}

uint32_t** GetDeviceRenderBufferAddress()
{
	uint32_t** buffer;
	CheckCudaErrors(hipGetSymbolAddress((void**)&buffer, renderBuffer));
	return buffer;
}

uint32_t* GetDeviceFrameNumberAddress()
{
	uint32_t* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, frameNumber));
	return target;
}

uint32_t* GetDeviceBounceAddress()
{
	uint32_t* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, bounce));
	return target;
}

D_BVH8* GetDeviceTLASAddress()
{
	D_BVH8* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, tlas));
	return target;
}

D_BVH8** GetDeviceBVHAddress()
{
	D_BVH8** target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, bvhs));
	return target;
}

D_BVHInstance** GetDeviceBLASAddress()
{
	D_BVHInstance** target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, blas));
	return target;
}

D_PathStateSAO* GetDevicePathStateAddress()
{
	D_PathStateSAO* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, pathState));
	return target;
}

D_ShadowTraceRequestSAO* GetDeviceShadowTraceRequestAddress()
{
	D_ShadowTraceRequestSAO* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, shadowTraceRequest));
	return target;
}

D_TraceRequestSAO* GetDeviceTraceRequestAddress()
{
	D_TraceRequestSAO* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, traceRequest));
	return target;
}

D_MaterialRequestSAO* GetDeviceDiffuseRequestAddress()
{
	D_MaterialRequestSAO* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, diffuseMaterialBuffer));
	return target;
}

D_MaterialRequestSAO* GetDevicePlasticRequestAddress()
{
	D_MaterialRequestSAO* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, plasticMaterialBuffer));
	return target;
}

D_MaterialRequestSAO* GetDeviceDielectricRequestAddress()
{
	D_MaterialRequestSAO* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, dielectricMaterialBuffer));
	return target;
}

D_MaterialRequestSAO* GetDeviceConductorRequestAddress()
{
	D_MaterialRequestSAO* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, conductorMaterialBuffer));
	return target;
}

D_QueueSize* GetDeviceQueueSizeAddress()
{
	D_QueueSize* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, queueSize));
	return target;
}

D_PixelQuery* GetDevicePixelQueryAddress()
{
	D_PixelQuery* target;
	CheckCudaErrors(hipGetSymbolAddress((void**)&target, pixelQuery));
	return target;
}
